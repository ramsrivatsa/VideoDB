#include "GPUUtils.h"

using namespace ucw;

int getNumGPUs() {
    int numGpus = 0;
    hipGetDeviceCount(&numGpus);
    return numGpus;
}

void setGPU(int id) {
    hipSetDevice(id);
}

bool hasCuda() {
    return true;
}