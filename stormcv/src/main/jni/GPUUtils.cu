#include "GPUUtils.h"

int getNumGPUs() {
    int numGpus = 0;
    hipGetDeviceCount(&numGpus);
    return numGpus;
}

void setGPU(int id) {
    hipSetDevice(id);
}
