#include "GPUUtils.h"

int getNumGPUs() {
    int numGpus = 0;
    hipGetDeviceCount(&numGpus);
    return numGpus;
}

void setGpuDevice(int id) {
    hipSetDevice(id);
}

bool hasCuda() {
    return true;
}